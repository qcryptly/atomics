#include "hip/hip_runtime.h"
#include <iostream>
#include "cryptly/cuda_ops.hxx"

namespace cryptly {
inline namespace v1 {

int _r(int err) {
	if (err != hipSuccess) {
		std::cout << "Error: " << hipGetErrorString(hipError_t(err)) << std::endl;
	}
	return err;
}

int print_devices() {
	int n_devices{};
	hipError_t err = hipGetDeviceCount(&n_devices);
	if (err != hipSuccess) return _r(err);

	std::cout << "Number of devices we have: " << n_devices << std::endl;
	for (int i = 0; i < n_devices; i++) {
		hipDeviceProp_t prop{};
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device name: " << prop.name << std::endl;
		std::cout << "Memory clock rate (KHz): " << prop.memoryClockRate << std::endl;
		std::cout << "Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
		std::cout << "Peak Memory Bandwidth (GB/s): " << peak_mem_band(
			prop.memoryBusWidth, prop.memoryClockRate);
		std::cout << "Max Grid Size" << prop.maxGridSize << std::endl; 
		std::cout << "Max Threads Per Block" << prop.maxThreadsPerBlock << std::endl; 
		std::cout << "Max Threads Per Dim" << prop.maxThreadsDim << std::endl; 
		std::cout << "Warp Size" << prop.warpSize << std::endl; 
	}
	return hipSuccess;
};

}
}
