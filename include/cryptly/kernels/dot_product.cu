#include "hip/hip_runtime.h"
#include "cryptly/cuda_ops.hxx"
#include "cryptly/kernels/dot_product.hxx"

template<int BSize = 256, class TArg>
__global__
void dot_product_cuda(std::size_t N, const TArg* a, const TArg* b, TArg* r){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	__shared__ TArg agg[BSize];
	agg[threadIdx.x] = 0;
	// Map
	for (int i = index; i < N; i += stride) {
		agg[threadIdx.x] += a[i] * b[i];
	}

	// Wait for all block threads to finish
	__syncthreads();

	// Reduce everything
	// Use one writer,
	// warp divergence
	if (threadIdx.x == 0) {
		r[blockIdx.x] = 0; 
		for (int i = 0; i < blockDim.x; i ++) {
			r[blockIdx.x] += agg[i];
		}
	}
}

template<class T, typename... Ts>
int dot_product_impl<T, Ts...>::operator()(const kernel_ps_t& ps,
	std::size_t N, const T* a_i, const T* b_i, T* r_o) {
	T *cuda_mem_a, *cuda_mem_b, *cuda_mem_r;
	auto size_n = sizeof(T) * N;
	auto size_r = sizeof(T) * ps.grid_size;
	// Allocate on Device
	_r(hipMalloc(&cuda_mem_a, size_n));	
	_r(hipMalloc(&cuda_mem_b, size_n));	
	_r(hipMalloc(&cuda_mem_r, size_r));	

	// Copy Host to Device
	_r(hipMemcpy(cuda_mem_a, a_i, size_n, hipMemcpyHostToDevice));
	_r(hipMemcpy(cuda_mem_b, b_i, size_n, hipMemcpyHostToDevice));

	// Run Kernel
	// Hard code grid size for now
	dot_product_cuda<256, T><<<ps.grid_size, ps.block_size, size_r, (ihipStream_t*)ps.hip_stream>>>(N, cuda_mem_a, cuda_mem_b, cuda_mem_r);	
	
	hipDeviceSynchronize();
	// Copy Device to Host
	T results[ps.grid_size];
	_r(hipMemcpy(results, cuda_mem_r, size_r, hipMemcpyDeviceToHost));

	// Simple aggregation
	T product{};
	for(int i = 0; i < ps.grid_size; i++) {
		product += results[i];
	}
	// Serialize and print results
	hipFree(cuda_mem_a);
	hipFree(cuda_mem_b);
	hipFree(cuda_mem_r);
	*r_o = product;
	return hipGetLastError();
}

// This is a hack to force the compiler
// to instantiate add_impl. If you leave
// this out, you will get undefined symbol
// reference for add_impl member functions :(
template <typename TArg, typename... Ts>
void __hack_impl_dot_product__(){
	[[maybe_unused]]TArg* _ = nullptr;
	(dot_product{})(
		kernel_ps_t{}, std::size_t{},
		_, _, _); 
	__hack_impl_dot_product__<Ts...>();
}

template<>
void __hack_impl_dot_product__<bool>(){
	[[maybe_unused]]bool* _ = nullptr;
	(dot_product{})(
		kernel_ps_t{}, std::size_t{},
		_, _, _);
}

void __hack_dot_product__(){
	__hack_impl_dot_product__<primitive_types>();
}
