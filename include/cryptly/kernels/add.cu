#include "hip/hip_runtime.h"
#include "cryptly/cuda_ops.hxx"
#include "cryptly/kernels/add.hxx"

namespace cryptly {
inline namespace v1 {

// Kernel function to add the elements of two arrays
template<class TArg=int>
__global__
void add_cuda(std::size_t n, TArg *a, TArg *b, TArg *r)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    r[i] = a[i] + b[i];
}

template<typename T, typename... Ts>
int add_impl<T, Ts...>::operator()(const kernel_ps_t& ps,
	std::size_t N, const T* a_i, const T* b_i, T* r_o){
	hipError_t error{};
	T *cuda_mem_a, *cuda_mem_b, *cuda_mem_r;
	auto n_bytes = N * sizeof(T);
	// _r logs any cuda errors, can turn this off
	// in production
	_r(hipMalloc(&cuda_mem_a, n_bytes));
	_r(hipMalloc(&cuda_mem_b, n_bytes));
	_r(hipMalloc(&cuda_mem_r, n_bytes));
	// Copy bytes from a_i and b_i
	_r(hipMemcpy(cuda_mem_a, a_i, n_bytes, hipMemcpyHostToDevice));	
	_r(hipMemcpy(cuda_mem_b, b_i, n_bytes, hipMemcpyHostToDevice));	
	add_cuda<T><<<ps.grid_size, ps.block_size, ps.shared_size, (ihipStream_t*)ps.hip_stream>>>(N, cuda_mem_a, cuda_mem_b, cuda_mem_r);
	hipDeviceSynchronize();
	_r(hipMemcpy(r_o, cuda_mem_r, n_bytes, hipMemcpyDeviceToHost));	
	hipFree(cuda_mem_a);
	hipFree(cuda_mem_b);
	return error;	
}

// This is a hack to force the compiler
// to instantiate add_impl. If you leave
// this out, you will get undefined symbol
// reference for add_impl member functions :(
template <typename TArg, typename... Ts>
void __hack_impl__(){
	[[maybe_unused]]TArg* _ = nullptr;
	(add{})(
		kernel_ps_t{}, std::size_t{},
		_, _, _); 
	__hack_impl__<Ts...>();
}

template<>
void __hack_impl__<bool>(){
	[[maybe_unused]]bool* _ = nullptr;
	(add{})(
		kernel_ps_t{}, std::size_t{},
		_, _, _);
}

void __hack__(){
	__hack_impl__<primitive_types>();
}

}
}
